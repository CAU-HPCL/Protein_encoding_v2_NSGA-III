#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include <python3.10/Python.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "../include/common.cuh"
#include "../include/info.cuh"
#include "../include/utils.cuh"
#include "../include/mutation.cuh"
#include "../include/sorting.cuh"

#define _CRT_SECURE_NO_WARINGS

using namespace cooperative_groups;

__global__ void mainKernel(hiprandStateXORWOW *random_generator, unsigned long long seed, const char *d_amino_seq_idx, char *d_population, float *d_obj_val, char *d_obj_idx, int *d_pql, char *d_tmp_population, float *d_tmp_obj_val, char *d_tmp_obj_idx, int *d_tmp_pql, int *d_sorted_array, bool *F_set, bool *Sp_set, int *d_np, int *d_rank_count)
{
    auto g = this_grid();
    auto tb = this_thread_block();
    hiprand_init(seed, g.thread_rank(), 0, &random_generator[g.thread_rank()]);

    /* Shared memory allocation */
    extern __shared__ int smem[];
    __shared__ int *s_pql;
    __shared__ int *s_mutex;
    __shared__ int *s_proceed_check;
    __shared__ int *s_termination_check;
    __shared__ float *s_obj_val;
    __shared__ float *s_obj_buffer;
    __shared__ char *s_amino_seq_idx;
    __shared__ char *s_solution;
    __shared__ char *s_obj_idx;
    __shared__ char *s_mutation_type;

    s_pql = smem;
    s_mutex = (int *)&s_pql[3];
    s_proceed_check = (int *)&s_mutex[1];
    s_termination_check = (int *)&s_proceed_check[1];
    s_obj_val = (float *)&s_termination_check[1];
    s_obj_buffer = (float *)&s_obj_val[OBJECTIVE_NUM];
    s_amino_seq_idx = (char *)&s_obj_buffer[tb.size()];
    s_solution = (char *)&s_amino_seq_idx[c_amino_seq_len];
    s_obj_idx = (char *)&s_solution[c_solution_len];
    s_mutation_type = (char *)&s_obj_idx[OBJECTIVE_NUM * 2];

    int partition_num;
    hiprandStateXORWOW local_generator = random_generator[g.thread_rank()];
    partition_num = (c_amino_seq_len % tb.size() == 0) ? (c_amino_seq_len / tb.size()) : (c_amino_seq_len / tb.size()) + 1;
    for (int i = 0; i < partition_num; i++)
    {
        int idx = tb.size() * i + tb.thread_rank();
        if (idx < c_amino_seq_len)
        {
            s_amino_seq_idx[idx] = d_amino_seq_idx[idx];
        }
    }
    tb.sync();

    int cycle_partition_num;
    int i, j;
    cycle_partition_num = (c_N % g.num_blocks() == 0) ? (c_N / g.num_blocks()) : (c_N / g.num_blocks()) + 1;

    /* Solutions initialization */
    for (i = 0; i < cycle_partition_num; i++)
    {
        int solution_idx = g.num_blocks() * i + g.block_rank();
        if (solution_idx < c_N)
        {
            if (solution_idx == (c_N - 1))
            {
                genPopulation(tb, &local_generator, s_amino_seq_idx, s_solution, HIGHEST_CAI_GEN);
            }
            else
            {
                genPopulation(tb, &local_generator, s_amino_seq_idx, s_solution, RANDOM_GEN);
            }

            /* Calculating objective function (+ 논문에 따라 추가적인 정규화 작업이 필요할 수 있음)*/
            calMinimumCAI(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
            calMinimumCBP(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
            calMinimumHSC(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
            calMinimumHD(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
            calMaximumGC(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
            calMaximumSL(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx, s_pql, s_mutex);

            copySolution(tb, s_solution, s_obj_val, s_obj_idx, s_pql, &d_population[c_solution_len * solution_idx], &d_obj_val[OBJECTIVE_NUM * solution_idx], &d_obj_idx[OBJECTIVE_NUM * 2 * solution_idx], &d_pql[3 * solution_idx]);  
            copySolution(tb, s_solution, s_obj_val, s_obj_idx, s_pql, &d_tmp_population[c_solution_len * solution_idx], &d_tmp_obj_val[OBJECTIVE_NUM * solution_idx], &d_tmp_obj_idx[OBJECTIVE_NUM * 2 * solution_idx], &d_tmp_pql[3 * solution_idx]);
            d_sorted_array[solution_idx] = solution_idx;
        }
    }
    g.sync();

    /* Starting Generation */
    for (i = 0; i < c_gen_cycle_num; i++)
    {
        for (j = 0; j < cycle_partition_num; j++)
        {
            int solution_idx = g.num_blocks() * j + g.block_rank();
            if (solution_idx < c_N)
            {
                if (i != 0)
                {
                    copySolution(tb, &d_tmp_population[c_solution_len * d_sorted_array[solution_idx]], &d_tmp_obj_val[OBJECTIVE_NUM * d_sorted_array[solution_idx]], &d_tmp_obj_idx[OBJECTIVE_NUM * 2 * d_sorted_array[solution_idx]], &d_tmp_pql[3 * solution_idx], &d_population[c_solution_len * solution_idx], &d_obj_val[OBJECTIVE_NUM * solution_idx], &d_obj_idx[OBJECTIVE_NUM * 2 * solution_idx], &d_pql[3 * solution_idx]);
                    copySolution(tb, &d_population[c_solution_len * solution_idx], &d_obj_val[OBJECTIVE_NUM * solution_idx], &d_obj_idx[OBJECTIVE_NUM * 2 * solution_idx], &d_pql[3 * solution_idx], s_solution, s_obj_val, s_obj_idx, s_pql);
                }

                /* Mutation */
                if (tb.thread_rank() == 0)
                {
                    do
                    {
                        *s_mutation_type = (char)(hiprand_uniform(&local_generator) * (OBJECTIVE_NUM + 1));
                    } while (*s_mutation_type == (OBJECTIVE_NUM + 1));
                }
                tb.sync();

                switch (*s_mutation_type)
                {
                case 0:
                    mutationRandom(tb, &local_generator, s_solution, s_amino_seq_idx, s_obj_idx);
                    break;
                case 1:
                    mutationCAI(tb, &local_generator, s_solution, s_amino_seq_idx, s_obj_idx, SELECT_UPPER_RANDOM);
                    break;
                case 2:
                    mutationCBP(tb, &local_generator, s_solution, s_amino_seq_idx, s_obj_idx, SELECT_UPPER_RANDOM, i % 2);
                    break;
                case 3:
                    mutationHSC(tb, &local_generator, s_solution, s_amino_seq_idx, s_obj_idx, SELECT_UPPER_RANDOM, i % 2);
                    break;
                case 4:
                    mutationHD(tb, &local_generator, s_solution, s_amino_seq_idx, s_obj_idx);
                    break;
                case 5:
                    mutationGC(tb, &local_generator, s_solution, s_amino_seq_idx, s_obj_idx, SELECT_LOW_GC); // 여기는 low high 추가적인 조치가 필요함
                    break;
                case 6:
                    mutationSL(tb, &local_generator, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx, s_pql, s_mutex, s_proceed_check, s_termination_check);
                    break;
                }
                tb.sync();

                /* Calculating objective function (+ 논문에 따라 추가적인 정규화 작업이 필요할 수 있음) */
                calMinimumCAI(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
                calMinimumCBP(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
                calMinimumHSC(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
                calMinimumHD(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
                calMaximumGC(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx);
                calMaximumSL(tb, s_solution, s_amino_seq_idx, s_obj_buffer, s_obj_val, s_obj_idx, s_pql, s_mutex);

                copySolution(tb, s_solution, s_obj_val, s_obj_idx, s_pql, &d_population[c_solution_len * (c_N + solution_idx)], &d_obj_val[OBJECTIVE_NUM * (c_N + solution_idx)], &d_obj_idx[OBJECTIVE_NUM * 2 * (c_N + solution_idx)], &d_pql[3 * (c_N + solution_idx)]);
            }
        }
        g.sync();

        int copy_partition_num = ((c_N * 2) % g.num_blocks() == 0) ? ((c_N * 2) / g.num_blocks()) : ((c_N * 2) / g.num_blocks()) + 1;
        for (j = 0; j < copy_partition_num; j++)
        {
            int solution_idx = g.num_blocks() * j + g.block_rank();
            if (solution_idx < (c_N * 2))
            {
                copySolution(tb, &d_population[c_solution_len * solution_idx], &d_obj_val[OBJECTIVE_NUM * solution_idx], &d_obj_idx[OBJECTIVE_NUM * 2 * solution_idx], &d_pql[3 * solution_idx], &d_tmp_population[c_solution_len * solution_idx], &d_tmp_obj_val[OBJECTIVE_NUM * solution_idx], &d_tmp_obj_idx[OBJECTIVE_NUM * 2 * solution_idx], &d_tmp_pql[3 * solution_idx]);
            }
        }
        g.sync();

        /* Sorting */
        nonDominatedSorting(g, d_obj_val, d_sorted_array, F_set, Sp_set, d_np, d_rank_count);
    }

    /* Memory copy from shared memory to global memory */
    for (i = 0; i < cycle_partition_num; i++)
    {
        int solution_idx = g.num_blocks() * i + g.block_rank();
        if (solution_idx < c_N)
        {
            copySolution(tb, &d_tmp_population[c_solution_len * d_sorted_array[solution_idx]], &d_tmp_obj_val[OBJECTIVE_NUM * d_sorted_array[solution_idx]], &d_tmp_obj_idx[OBJECTIVE_NUM * 2 * d_sorted_array[solution_idx]], &d_tmp_pql[3 * d_sorted_array[solution_idx]], &d_population[c_solution_len * solution_idx], &d_obj_val[OBJECTIVE_NUM * solution_idx], &d_obj_idx[OBJECTIVE_NUM * 2 * solution_idx], &d_pql[3 * solution_idx]);
        }
    }

    return;
}

/*
argv[1] : Input file name
argv[2] : Population size (N)
argv[3] : Generation count (G)
argv[4] : Number of CDS
argv[5] : Mutation probability (Pm)
argv[6] : Number of threads per block

For example
../Protein_FASTA/Q5VZP5.fasta.txt  10 10 2 0.5 32
*/
int main(const int argc, const char *argv[])
{
    srand((unsigned int)time(NULL));

    /* Getting information of Deivce */
    hipDeviceProp_t deviceProp;
    int dev = 0;
    int maxSharedMemPerBlock;
    int maxSharedMemPerProcessor;
    int totalConstantMem;
    int maxRegisterPerProcessor;
    int maxRegisterPerBlock;
    int totalMultiProcessor;

    CHECK_CUDA(hipGetDeviceProperties(&deviceProp, dev))
    CHECK_CUDA(hipDeviceGetAttribute(&maxSharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, dev))
    CHECK_CUDA(hipDeviceGetAttribute(&maxSharedMemPerProcessor, hipDeviceAttributeMaxRegistersPerMultiprocessor, dev))
    CHECK_CUDA(hipDeviceGetAttribute(&totalConstantMem, hipDeviceAttributeTotalConstantMemory, dev))
    CHECK_CUDA(hipDeviceGetAttribute(&maxRegisterPerProcessor, hipDeviceAttributeMaxRegistersPerMultiprocessor, dev))
    CHECK_CUDA(hipDeviceGetAttribute(&maxRegisterPerBlock, hipDeviceAttributeMaxRegistersPerBlock, dev))
    CHECK_CUDA(hipDeviceGetAttribute(&totalMultiProcessor, hipDeviceAttributeMultiprocessorCount, dev))

    printf("Device #%d:\n", dev);
    printf("Name: %s\n", deviceProp.name);
    printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Clock rate: %d MHz\n", deviceProp.clockRate / 1000);
    printf("Global memory size: %lu MB\n", deviceProp.totalGlobalMem / (1024 * 1024));
    printf("Max thread dimensions: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("Max grid dimensions: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("Total constant memory: %d bytes\n", totalConstantMem);
    printf("Max threads per SM: %d\n", deviceProp.maxThreadsPerMultiProcessor);
    printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Maximum shared memory per SM: %d bytes\n", maxSharedMemPerProcessor);
    printf("Maximum shared memory per block: %d bytes\n", maxSharedMemPerBlock);
    printf("Maximum number of registers per SM: %d\n", maxRegisterPerProcessor);
    printf("Maximum number of registers per block: %d\n", maxRegisterPerBlock);
    printf("Total number of SM in device: %d\n", totalMultiProcessor);
    printf("\n");

    /* Checking input parameters */
    int population_size = atoi(argv[2]);
    int gen_cycle_num = atoi(argv[3]);
    char cds_num = (char)atoi(argv[4]);
    float mutation_prob = atof(argv[5]);
    int threads_per_block = atoi(argv[6]);
    if ((population_size <= 0) || (gen_cycle_num < 0) || (cds_num <= 1) || (mutation_prob < 0.f) || (mutation_prob > 1.f))
    {
        printf("Line : %d Please cheking input parameters.. \n", __LINE__);
        return EXIT_FAILURE;
    }

    /* Preprocessing */
    FILE *fp;
    char buffer[256];
    char *amino_seq;
    int amino_seq_len, cds_len, solution_len;
    int idx;

    fp = fopen(argv[1], "r");
    if (fp == NULL)
    {
        printf("Line : %d Opening Protein FASTA format file is failed.. \n", __LINE__);
        return EXIT_FAILURE;
    }
    fseek(fp, 0, SEEK_END);
    amino_seq_len = ftell(fp);
    fseek(fp, 0, SEEK_SET);
    fgets(buffer, 256, fp);
    amino_seq_len -= ftell(fp);
    amino_seq = (char *)malloc(sizeof(char) * (amino_seq_len + 1)); // +1 indicates last is stop codons.

    idx = 0;
    while (!feof(fp))
    {
        char tmp = fgetc(fp);
        if (tmp != '\n')
        {
            amino_seq[idx++] = tmp;
        }
    }
    amino_seq[idx - 1] = 'Z';
    amino_seq[idx] = (char)NULL;
    amino_seq_len = idx;
    cds_len = amino_seq_len * CODON_SIZE;
    solution_len = cds_len * cds_num;
    fclose(fp);

    unsigned long long seed = (unsigned long long)rand();
    char *h_amino_seq_idx;
    char *h_population;
    float *h_obj_val;
    char *h_obj_idx; // 나중에 제거
    float *h_reference_points;

    hiprandStateXORWOW *d_random_generator;
    hipEvent_t d_start, d_end;
    unsigned long long *d_seed;
    char *d_amino_seq_idx;
    char *d_population;
    float *d_obj_val;
    char *d_obj_idx;
    int *d_pql;

    // sorting 을 위해서 할당한 것들
    int *d_np;
    bool *d_F_set, *d_Sp_set;
    int *d_rank_count;
    int *d_sorted_array;
    char *d_tmp_population;
    float *d_tmp_obj_val;
    char *d_tmp_obj_idx;
    int *d_tmp_pql;

    h_amino_seq_idx = (char *)malloc(sizeof(char) * amino_seq_len);
    for (int i = 0; i < amino_seq_len; i++)
    {
        h_amino_seq_idx[i] = findAminoIndex(amino_seq[i]);
    }

    /* Setting Reference points */
    h_reference_points = (float *)malloc(sizeof(float) * OBJECTIVE_NUM * population_size);
    getReferencePoints(h_reference_points, OBJECTIVE_NUM, population_size);

    int blocks_num;
    int numBlocksPerSm = 0;
    size_t using_shared_memory_size = sizeof(float) * (OBJECTIVE_NUM + threads_per_block) + sizeof(char) * (amino_seq_len + solution_len + (OBJECTIVE_NUM * 2) + 1) + sizeof(int) * 6;
    size_t using_constant_memory_size = sizeof(codons_start_idx) + sizeof(syn_codons_num) + sizeof(codons) + sizeof(codons_weight) + sizeof(cps) + sizeof(int) * 5 + sizeof(char) + sizeof(float);

    CHECK_CUDA(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, mainKernel, threads_per_block, using_shared_memory_size))
    if (population_size > (deviceProp.multiProcessorCount * numBlocksPerSm))
    {
        blocks_num = (deviceProp.multiProcessorCount * numBlocksPerSm);
    }
    else
    {
        blocks_num = population_size;
    }
    // TODO: 나중에 다시 계산 필요한 부분
    size_t using_global_memory_size = sizeof(hiprandStateXORWOW) * (blocks_num * threads_per_block) + sizeof(unsigned long long) + sizeof(char) * (amino_seq_len + solution_len * population_size * 2 + OBJECTIVE_NUM * 2 * population_size * 2) + sizeof(float) * (OBJECTIVE_NUM * population_size * 2);

    /* Host Memory allocation */
    h_population = (char *)malloc(sizeof(char) * solution_len * population_size * 2);
    h_obj_val = (float *)malloc(sizeof(float) * OBJECTIVE_NUM * population_size * 2);
    h_obj_idx = (char *)malloc(sizeof(char) * OBJECTIVE_NUM * 2 * population_size * 2); // 나중에 제거

    /* Device Memory allocation */
    CHECK_CUDA(hipEventCreate(&d_start))
    CHECK_CUDA(hipEventCreate(&d_end))
    CHECK_CUDA(hipMalloc((void **)&d_random_generator, sizeof(hiprandStateXORWOW) * blocks_num * threads_per_block))
    CHECK_CUDA(hipMalloc((void **)&d_seed, sizeof(unsigned long long)))
    CHECK_CUDA(hipMalloc((void **)&d_amino_seq_idx, sizeof(char) * amino_seq_len))
    CHECK_CUDA(hipMalloc((void **)&d_population, sizeof(char) * solution_len * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_obj_val, sizeof(float) * OBJECTIVE_NUM * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_obj_idx, sizeof(char) * OBJECTIVE_NUM * 2 * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_pql, sizeof(int) * 3 * population_size * 2))

    CHECK_CUDA(hipMalloc((void **)&d_tmp_population, sizeof(char) * solution_len * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_tmp_obj_val, sizeof(float) * OBJECTIVE_NUM * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_tmp_obj_idx, sizeof(char) * OBJECTIVE_NUM * 2 * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_tmp_pql, sizeof(int) * 3 * population_size * 2))

    CHECK_CUDA(hipMalloc((void **)&d_sorted_array, sizeof(int) * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_rank_count, sizeof(int) * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_np, sizeof(int) * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_F_set, sizeof(bool) * population_size * 2 * population_size * 2))
    CHECK_CUDA(hipMalloc((void **)&d_Sp_set, sizeof(bool) * population_size * 2 * population_size * 2))

    /* Memory copy Host to Device */
    CHECK_CUDA(hipMemcpy(d_seed, &seed, sizeof(unsigned long long), hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(d_amino_seq_idx, h_amino_seq_idx, sizeof(char) * amino_seq_len, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons_start_idx), codons_start_idx, sizeof(codons_start_idx)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_syn_codons_num), syn_codons_num, sizeof(syn_codons_num)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons), codons, sizeof(codons)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons_weight), codons_weight, sizeof(codons_weight)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_cps), cps, sizeof(cps)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_N), &population_size, sizeof(int)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_amino_seq_len), &amino_seq_len, sizeof(int)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_solution_len), &solution_len, sizeof(int)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_cds_len), &cds_len, sizeof(int)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_cds_num), &cds_num, sizeof(char)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_mutation_prob), &mutation_prob, sizeof(float)))
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_gen_cycle_num), &gen_cycle_num, sizeof(int)))

    /* CUDA Kerenl call */
    printf("Global memory usage : %lu bytes\n", using_global_memory_size);
    printf("Shared memory usage : %lu bytes\n", using_shared_memory_size);
    printf("Constant memory usage : %lu bytes\n", using_constant_memory_size);
    float kernel_time = 0.f;
    void *args[] = {&d_random_generator, &d_seed, &d_amino_seq_idx, &d_population, &d_obj_val, &d_obj_idx, &d_pql, &d_tmp_population, &d_tmp_obj_val, &d_tmp_obj_idx, &d_tmp_pql, &d_sorted_array, &d_F_set, &d_Sp_set, &d_np, &d_rank_count};
    CHECK_CUDA(hipEventRecord(d_start))
    CHECK_CUDA(hipLaunchCooperativeKernel((void **)mainKernel, blocks_num, threads_per_block, args, using_shared_memory_size))
    CHECK_CUDA(hipDeviceSynchronize())
    CHECK_CUDA(hipEventRecord(d_end))
    CHECK_CUDA(hipEventSynchronize(d_end))
    CHECK_CUDA(hipEventElapsedTime(&kernel_time, d_start, d_end))
    kernel_time /= 1000.f;
    printf("Kernel time : %f\n", kernel_time);

    /* Memory copy Device to Host */
    CHECK_CUDA(hipMemcpy(h_population, d_population, sizeof(char) * solution_len * population_size * 2, hipMemcpyDeviceToHost))
    CHECK_CUDA(hipMemcpy(h_obj_val, d_obj_val, sizeof(float) * OBJECTIVE_NUM * population_size * 2, hipMemcpyDeviceToHost))
    CHECK_CUDA(hipMemcpy(h_obj_idx, d_obj_idx, sizeof(char) * OBJECTIVE_NUM * 2 * population_size * 2, hipMemcpyDeviceToHost))

    /* Print */
    for (int i = 0; i < population_size; i++)
    {
        for (int j = 0; j < cds_num; j++)
        {
            for (int k = 0; k < cds_len; k++)
            {
                printf("%c", h_population[i * solution_len + cds_len * j + k]);
            }
            printf("\n");
        }
        printf("\n %d mCAI : %f\n", h_obj_idx[i * OBJECTIVE_NUM * 2 + MIN_CAI_IDX * 2], h_obj_val[i * OBJECTIVE_NUM + MIN_CAI_IDX]);
        printf("\n %d mCBP : %f\n", h_obj_idx[i * OBJECTIVE_NUM * 2 + MIN_CBP_IDX * 2], h_obj_val[i * OBJECTIVE_NUM + MIN_CBP_IDX]);
        printf("\n %d mHSC : %f\n", h_obj_idx[i * OBJECTIVE_NUM * 2 + MIN_HSC_IDX * 2], h_obj_val[i * OBJECTIVE_NUM + MIN_HSC_IDX]);
        printf("\n %d  %d mHD : %f\n", h_obj_idx[i * OBJECTIVE_NUM * 2 + MIN_HD_IDX * 2], h_obj_idx[i * OBJECTIVE_NUM * 2 + MIN_HD_IDX * 2 + 1], h_obj_val[i * OBJECTIVE_NUM + MIN_HD_IDX]);
        printf("\n %d MGC : %f\n", h_obj_idx[i * OBJECTIVE_NUM * 2 + MAX_GC_IDX * 2], h_obj_val[i * OBJECTIVE_NUM + MAX_GC_IDX]);
        printf("\n %d MSL : %f\n", h_obj_idx[i * OBJECTIVE_NUM * 2 + MAX_SL_IDX * 2], h_obj_val[i * OBJECTIVE_NUM + MAX_SL_IDX]);
        printf("\n");
    }

    /* free host memory */
    free(amino_seq);
    free(h_amino_seq_idx);
    free(h_population);
    free(h_obj_val);
    free(h_obj_idx); // 나중에 제거
    free(h_reference_points);

    /* free deivce memory */
    CHECK_CUDA(hipEventDestroy(d_start))
    CHECK_CUDA(hipEventDestroy(d_end))
    CHECK_CUDA(hipFree(d_random_generator))
    CHECK_CUDA(hipFree(d_seed))
    CHECK_CUDA(hipFree(d_amino_seq_idx))
    CHECK_CUDA(hipFree(d_population))
    CHECK_CUDA(hipFree(d_obj_val))
    CHECK_CUDA(hipFree(d_obj_idx))
    CHECK_CUDA(hipFree(d_pql))

    CHECK_CUDA(hipFree(d_tmp_population))
    CHECK_CUDA(hipFree(d_tmp_obj_val))
    CHECK_CUDA(hipFree(d_tmp_obj_idx))
    CHECK_CUDA(hipFree(d_tmp_pql))
    CHECK_CUDA(hipFree(d_sorted_array))
    CHECK_CUDA(hipFree(d_np))
    CHECK_CUDA(hipFree(d_F_set))
    CHECK_CUDA(hipFree(d_Sp_set))
    CHECK_CUDA(hipFree(d_rank_count))

    return EXIT_SUCCESS;
}